#include "hip/hip_runtime.h"
#include "PollardsRhoCuda.h"
#include "gmp.h"
#include <iostream>
#include "include/cgbn/cgbn.h"
#include "utility/cpu_support.h"
#include "utility/gpu_support.h"

#define TPI 8
#define BITS 32*8

typedef cgbn_context_t<TPI> context_t;
typedef cgbn_env_t<context_t, BITS> env_t;


typedef struct{
    cgbn_mem_t<BITS> to_factor;
    cgbn_mem_t<BITS> start_point;
    cgbn_mem_t<BITS> a;
    bool is_factored;
} PollardStart;


__device__ bool factored = false;


__global__ void TryPollardRho(PollardStart* starting) {
    uint32_t instance = (blockIdx.x*blockDim.x + threadIdx.x) / TPI;

    context_t      bn_context(cgbn_no_checks);
    env_t          bn_env(bn_context.env<env_t>());
    
    env_t::cgbn_t to_factor, diff, point, point2, a, product;

    cgbn_load(bn_env, to_factor, &(starting[instance].to_factor));
    cgbn_load(bn_env, point, &(starting[instance].start_point));
    cgbn_set(bn_env, point2, point);
    cgbn_load(bn_env, a, &(starting->a));
    cgbn_set_ui32(bn_env, product, 1);

    uint32_t mont_c = -cgbn_binary_inverse_ui32(bn_env, cgbn_get_ui32(bn_env, to_factor));
    int count = 0;

    while(!factored){
        cgbn_mont_sqr(bn_env, point, point, to_factor, mont_c);
        cgbn_add(bn_env, point, point, a);
        cgbn_mont_sqr(bn_env, point2, point2, to_factor, mont_c);
        cgbn_add(bn_env, point2, point2, a);
        cgbn_mont_sqr(bn_env, point2, point2, to_factor, mont_c);
        cgbn_add(bn_env, point2, point2, a);


        if(cgbn_compare(bn_env, point, point2) < 0){
            cgbn_sub(bn_env, diff, point2, point);
        }
        else{
            cgbn_sub(bn_env, diff, point, point2);
        }
        
        if(cgbn_compare_ui32(bn_env, diff, 0) == 0){
            return;
        }

        cgbn_mul(bn_env, product, product, diff);
        cgbn_rem(bn_env, product, product, to_factor);

        count++;
        if(count == 100){
            count = 0;
            cgbn_gcd(bn_env, product, product, to_factor);//here product is the factor
            
            if(cgbn_compare_ui32(bn_env, product, 1) > 0 && cgbn_compare(bn_env, product, to_factor) < 0){
                factored = true;
                cgbn_store(bn_env, &(starting[instance].start_point), product);
                starting[instance].is_factored = true;
            }
            cgbn_set_ui32(bn_env, product, 1);
        }
    }
}


void PollardsRhoCuda(mpz_t output, mpz_t to_factor, int thread_count){
    if(thread_count == 0){
        thread_count = 2048;
    }

    int problem_instances = thread_count;
    int inst_size = problem_instances * TPI;
    const int block_size = 512;
    int block_num = inst_size / block_size;

    const bool initial_factored = false;
    
    PollardStart instance_local[problem_instances];
    PollardStart* instance_cuda;

    mpz_t temp_random;
    mpz_init(temp_random);

    long seed = rand();
    gmp_randstate_t random_state;
    gmp_randinit_default(random_state);
    gmp_randseed_ui(random_state, seed);

    CUDA_CHECK(hipSetDevice(0));

    for(int i = 0; i < problem_instances; i++){
        PollardStart start;
        mpz_urandomm(temp_random, random_state, to_factor);
        from_mpz(temp_random, start.a._limbs, BITS/32);
        mpz_urandomm(temp_random, random_state, to_factor);
        from_mpz(temp_random, start.start_point._limbs, BITS/32);
        from_mpz(to_factor, start.to_factor._limbs, BITS/32);
        start.is_factored = false;
        instance_local[i] = start;
    }

    CUDA_CHECK(hipMalloc(&instance_cuda, problem_instances * sizeof(PollardStart)));
    CUDA_CHECK(hipMemcpyToSymbol(HIP_SYMBOL(factored), &initial_factored, sizeof(bool)));
    CUDA_CHECK(hipMemcpy(instance_cuda, &instance_local,  problem_instances * sizeof(PollardStart), hipMemcpyHostToDevice));

    TryPollardRho<<<block_num, block_size>>>(instance_cuda);

    hipMemcpy(&instance_local, instance_cuda, problem_instances*sizeof(PollardStart), hipMemcpyDeviceToHost);
    hipFree(&instance_cuda);

    for(int i = 0; i < problem_instances; i++){
        if(instance_local[i].is_factored){
            to_mpz(output, instance_local[i].start_point._limbs, BITS/32);
            break;
        }
    }

    mpz_clear(temp_random);
}
