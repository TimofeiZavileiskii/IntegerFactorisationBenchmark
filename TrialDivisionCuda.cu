#include "hip/hip_runtime.h"
#include "TrialDivisionCuda.h"
#include <gmp.h>
#include <iostream>
#include "include/cgbn/cgbn.h"
#include "utility/cpu_support.h"
#include "utility/gpu_support.h"

#define TPI 4
#define BITS 128

typedef cgbn_context_t<TPI> context_t;
typedef cgbn_env_t<context_t, BITS> env_t;


__device__ uint32_t range_size = 1;
__device__ bool factored = false;


typedef struct {
  cgbn_mem_t<BITS> to_factor;
  cgbn_mem_t<BITS> output;
} instance_t;


__global__ void try_division(instance_t* problem) {
    int32_t instance = (blockIdx.x*blockDim.x + threadIdx.x) / TPI;

    context_t      bn_context(cgbn_no_checks);
    env_t          bn_env(bn_context.env<env_t>());
    
    env_t::cgbn_t to_factor, range_start, try_mod, mod_out;

    cgbn_load(bn_env, to_factor, &(problem->to_factor));
    cgbn_set_ui32(bn_env, range_start, range_size);
    cgbn_mul_ui32(bn_env, range_start, range_start, instance);
    cgbn_add_ui32(bn_env, range_start, range_start, 2);

    if(instance == 0 && threadIdx.x % TPI == 0){
        printf(">> %i\n", range_size);
    }

    for(uint32_t i = 0; i < range_size && !factored; i++){
        cgbn_add_ui32(bn_env, try_mod, range_start, i);
        cgbn_rem(bn_env, mod_out, to_factor, try_mod);
        if(cgbn_compare_ui32(bn_env, mod_out, 0) == 0){
            env_t::cgbn_t other_factor;
            cgbn_div(bn_env, other_factor, to_factor, try_mod);
            
            if(cgbn_compare(bn_env, other_factor, try_mod) != -1){
                cgbn_store(bn_env, &(problem->output), try_mod);
            }
            else{
                cgbn_store(bn_env, &(problem->output), other_factor);
            }
            factored = true;
        }
    }
}


void TrialDivisionCuda(mpz_t& output, mpz_t& to_factor)
{
    const int num_kernels = 268435456;
    const int inst_size = num_kernels * TPI;
    const int block_size = 512;
    const int block_num = inst_size / block_size;
    
    const bool initial_factored = false;

    mpz_t bound, range_size_local;
    mpz_inits(bound, range_size_local, NULL);
    double bound_d = sqrt(mpz_get_d(to_factor));
    mpz_set_d(bound, bound_d);
    mpz_div_ui(range_size_local, bound, num_kernels);
    mpz_add_ui(range_size_local, range_size_local, 1);;
    
    uint32_t range_size_local_ui = mpz_get_ui(range_size_local);
    instance_t probelm_instance_local;
    instance_t* probelm_instance_cuda;

    CUDA_CHECK(hipSetDevice(0));

    CUDA_CHECK(hipMalloc(&probelm_instance_cuda, sizeof(instance_t)));

    from_mpz(to_factor, probelm_instance_local.to_factor._limbs, BITS/32);
    CUDA_CHECK(hipMemcpyToSymbol(HIP_SYMBOL(range_size), &range_size_local_ui, sizeof(uint32_t)));
    CUDA_CHECK(hipMemcpyToSymbol(HIP_SYMBOL(factored), &initial_factored, sizeof(bool)));

    CUDA_CHECK(hipMemcpy(probelm_instance_cuda, &probelm_instance_local, sizeof(instance_t), hipMemcpyHostToDevice));

    try_division<<<block_num, block_size>>>(probelm_instance_cuda);

    hipMemcpy(&probelm_instance_local, probelm_instance_cuda, sizeof(instance_t), hipMemcpyDeviceToHost);
    hipFree(&probelm_instance_cuda);
    to_mpz(output, probelm_instance_local.output._limbs, BITS/32);

    mpz_clears(bound, range_size_local, NULL);
}