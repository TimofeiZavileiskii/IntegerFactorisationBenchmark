#include "hip/hip_runtime.h"
#include "PollardsRhoCuda.h"
#include "EllipticCurves.h"
#include "gmp.h"
#include <iostream>
#include "include/cgbn/cgbn.h"
#include "utility/cpu_support.h"
#include "utility/gpu_support.h"
#include "ECM.h"
#include <stdio.h>

#include <algorithm>

#define TPI 16

const int BITS = 32*10;

#define BIT_COUNT_LONG(n) 64 - __clzll(n)
#define TEST_BIT(n, bit) ((n & (1 << bit)) != 0)

typedef cgbn_context_t<TPI> context_t;
typedef cgbn_env_t<context_t, BITS> env_t;


__device__ bool ecm_finished = false;


struct PointCuda{
    env_t::cgbn_t x;
    env_t::cgbn_t z;
    env_t::cgbn_t x_sub_z;
    env_t::cgbn_t x_add_z;
};


class CurveCuda{
    public:
    PointCuda point_curr;

    env_t::cgbn_t sqr1, sqr2, a_2_over_4, cross_1;
    env_t::cgbn_t mod, c;
    uint32_t montgomery_param;

    PointCuda point_copy;
    PointCuda point_double;

    context_t _context;
    env_t     _env;
    int32_t   _instance;


    __device__ __forceinline__ CurveCuda(cgbn_monitor_t monitor, cgbn_error_report_t *report, int32_t instance) : 
    _context(monitor, report, (uint32_t)instance), _env(_context), _instance(instance) {
    }


    __device__ __forceinline__ void InitMont(){
        montgomery_param = -cgbn_binary_inverse_ui32(_env, cgbn_get_ui32(_env, mod));
    }

    __device__ __forceinline__ void ModularMul(env_t::cgbn_t& c, const env_t::cgbn_t& a, const env_t::cgbn_t& b){
        #if m_arith == 1
            cgbn_mont_mul(_env, c, a, b, mod, montgomery_param);
        #else
            cgbn_mul(_env, c, a, b);
            cgbn_rem(_env, c, c, mod);
        #endif
    }

    __device__ __forceinline__ void ModularSqr(env_t::cgbn_t& c, const env_t::cgbn_t& a){
        #if m_arith == 1
            cgbn_mont_sqr(_env, c, a, mod, montgomery_param);
        #else
            cgbn_sqr(_env, c, a);
            cgbn_rem(_env, c, c, mod);
        #endif
    }

    __device__ __forceinline__ void ModularSub(env_t::cgbn_t& c, const env_t::cgbn_t& a, const env_t::cgbn_t& b){
        if(cgbn_compare(_env, a, b) < 0){
            cgbn_add(_env, c, a, mod);
            cgbn_sub(_env, c, c, b);
        }
        else{
            cgbn_sub(_env, c, a, b);
        }
    }

    __device__ __forceinline__ void ModularAdd(env_t::cgbn_t& c, const env_t::cgbn_t& a, const env_t::cgbn_t& b){
        cgbn_add(_env, c, a, mod);
        if(cgbn_compare(_env, c, mod) > 0){
            cgbn_sub(_env, c, c, mod); 
        }
    }

    __device__ __forceinline__ void PointComputeDiffCuda(PointCuda& point){
        ModularSub(point.x_sub_z, point.x, point.z);
        ModularAdd(point.x_add_z, point.x, point.z);
    }

    __device__ void CopyPointCuda(const PointCuda& point, PointCuda& point_copy){
        cgbn_set(_env, point_copy.x, point.x);
        cgbn_set(_env, point_copy.z, point.z);
        cgbn_set(_env, point_copy.x_add_z, point.x_add_z);
        cgbn_set(_env, point_copy.x_sub_z, point.x_sub_z);
    }

    /*
    inline void DoublePoint(MontgomeryPoint& point){
        MontgomeryMul(sqr1, point.x_add_z, point.x_add_z, params);
        MontgomeryMul(sqr2, point.x_sub_z, point.x_sub_z, params);
        ModularSub(cross_1, sqr1, sqr2, params.mod);
        MontgomeryMul(point.x, sqr1, sqr2, params);
        MontgomeryMul(point.z, a_2_over_4, cross_1, params);
        ModularAdd(point.z, point.z, sqr2, params.mod);
        MontgomeryMul(point.z, point.z, cross_1, params);
        point.ComputeDiff(params.mod);
    }*/

    __device__ void DoublePointCuda(PointCuda& point){
        ModularSqr(sqr1, point.x_add_z);
        ModularSqr(sqr2, point.x_sub_z);
        ModularSub(cross_1, sqr1, sqr2);
        ModularMul(point.x, sqr1, sqr2);
        ModularMul(point.z, a_2_over_4, cross_1);
        ModularAdd(point.z, point.z, sqr2);
        ModularMul(point.z, point.z, cross_1);
        PointComputeDiffCuda(point);
    }

    /*
    inline void AddPoints(MontgomeryPoint& point, MontgomeryPoint& to_add, MontgomeryPoint& p_min_add){
        if((mpz_cmp_ui(point.x, 0) == 0) && (mpz_cmp_ui(point.z, 0) == 0)){
            return;
        }
        if((mpz_cmp_ui(to_add.x, 0) == 0) && (mpz_cmp_ui(to_add.z, 0) == 0)){
            point.Copy(to_add);
            return;
        }

        #define cross_2 sqr1
        MontgomeryMul(cross_1, point.x_sub_z, to_add.x_add_z, params);
        MontgomeryMul(cross_2, point.x_add_z, to_add.x_sub_z, params);
        ModularAdd(to_add.x, cross_1, cross_2, params.mod);
        MontgomeryMul(to_add.x, to_add.x, to_add.x, params);
        MontgomeryMul(to_add.x, to_add.x, p_min_add.z, params);

        mpz_sub(to_add.z, cross_1, cross_2);
        MontgomeryMul(to_add.z, to_add.z, to_add.z, params);
        MontgomeryMul(to_add.z, to_add.z, p_min_add.x, params);
        to_add.ComputeDiff(params.mod);
        #undef cross_2
    }*/

    __device__ void AddPointsCuda(const PointCuda& point, PointCuda& to_add, const PointCuda& diff){
        /*
        if((cgbn_compare_ui32(_env, point.x, 0) == 0) && (cgbn_compare_ui32(_env, point.z, 0) == 0)){
            return;
        }
        if((cgbn_compare_ui32(_env, to_add.x, 0) == 0) && (cgbn_compare_ui32(_env, to_add.z, 0) == 0)){
            CopyPointCuda(point, to_add);
            return;
        }*/

        #define cross_2 sqr1
        ModularMul(cross_1, point.x_sub_z, to_add.x_add_z);
        ModularMul(cross_2, point.x_add_z, to_add.x_sub_z);
        ModularAdd(to_add.x, cross_1, cross_2);

        ModularSqr(to_add.x, to_add.x);
        ModularMul(to_add.x, to_add.x, diff.z);

        ModularSub(to_add.z, cross_1, cross_2);
        ModularSqr(to_add.z, to_add.z);
        ModularMul(to_add.z, to_add.z, diff.x);
        PointComputeDiffCuda(to_add);
        #undef cross_2
    }

    /*
    inline void MultPoints(MontgomeryPoint& point, long multiple){
        if(multiple == 0){
            mpz_set_ui(point.x, 0);
            mpz_set_ui(point.z, 0);
            return;
        }
        if(multiple == 1){
            return;
        }
        if(multiple == 2){
            DoublePoint(point);
            return;
        }
        
        point.Copy(p_copy);
        point.Copy(p_double);
        DoublePoint(p_double);
        
        int bit_count = (int)log2((double)multiple)+1;

        for(int i = bit_count-2; i > 0; i--){
            if(test_bit(multiple, i) == 1){
                AddPoints(p_double, p_copy, point);
                DoublePoint(p_double);
            }
            else{
                AddPoints(p_copy, p_double, point);
                DoublePoint(p_copy);
            }
        }
        if(test_bit(multiple, 0) == 1){
            AddPoints(p_double, p_copy, point);
            p_copy.Copy(point);
        }
        else{
            DoublePoint(p_copy);
            p_copy.Copy(point);
        }
    }
    */

    __device__ __forceinline__ void MultPointCuda(long multiple){
        if(multiple == 2){
            DoublePointCuda(point_curr);
            return;
        }

        CopyPointCuda(point_curr, point_copy);
        CopyPointCuda(point_curr, point_double);
        DoublePointCuda(point_double);

        int bit_count = BIT_COUNT_LONG(multiple);

        for(int i = bit_count-2; i > 0; i--){
            if(TEST_BIT(multiple, i)){
                AddPointsCuda(point_double, point_copy, point_curr);
                DoublePointCuda(point_double);
            }
            else{
                AddPointsCuda(point_copy, point_double, point_curr);
                DoublePointCuda(point_copy);
            }
            
        }

        if(TEST_BIT(multiple, 0)){
            AddPointsCuda(point_double, point_copy, point_curr);
            CopyPointCuda(point_copy, point_curr);
        }
        else{
            DoublePointCuda(point_copy);
            CopyPointCuda(point_copy, point_curr);
        }
    }
};


struct EcmStart {
    cgbn_mem_t<BITS> c;
    cgbn_mem_t<BITS> x;
    cgbn_mem_t<BITS> z;
    cgbn_mem_t<BITS> a_2_over_4;
    bool is_factored;
};


__global__ void EcmKernel(cgbn_error_report_t *report, EcmStart* start_instances, int inst_size, long* primes, cgbn_mem_t<BITS>* mod_host, int prime_count, long B1) {
    uint32_t instance = (blockIdx.x*blockDim.x + threadIdx.x) / TPI;
    if(instance >= inst_size){
        return;
    }

    env_t::cgbn_t gcd;

    CurveCuda curve(cgbn_report_monitor, report, instance);

    cgbn_load(curve._env, curve.mod, mod_host);
    cgbn_load(curve._env, curve.c, &(start_instances[instance].c));
    cgbn_load(curve._env, curve.point_curr.x, &(start_instances[instance].x));
    cgbn_load(curve._env, curve.point_curr.z, &(start_instances[instance].z));
    cgbn_load(curve._env, curve.a_2_over_4, &(start_instances[instance].a_2_over_4));

    curve.InitMont();
    curve.PointComputeDiffCuda(curve.point_curr);

    long e;

    float log_B1 = logf((float)B1);
    for(int index = 0; index < prime_count && !ecm_finished; index++){
        float prime = (float)primes[index];
        int repeat = (int)(log_B1/logf(prime));

        for(int rep = 0; rep < repeat; rep++)
            curve.MultPointCuda(prime);
    }
    
    cgbn_gcd(curve._env, gcd, curve.mod, curve.point_curr.z);
    cgbn_store(curve._env, &start_instances[instance].c, gcd);

    if(cgbn_compare_ui32(curve._env, gcd, 1) > 0 && cgbn_compare(curve._env, gcd, curve.mod) < 0){
        if(threadIdx.x % TPI == 0){}
        start_instances[instance].is_factored = true;
      //  cgbn_store(curve._env, &start_instances[instance].x, curve.point_curr.x);
       // cgbn_store(curve._env, &start_instances[instance].z, curve.point_curr.z);
        cgbn_store(curve._env, &start_instances[instance].c, gcd);
        ecm_finished = true;
    }
}

void EcmCuda(mpz_t output, mpz_t to_factor, int thread_count, const std::vector<long>& primes){
    if(thread_count == 0){
        thread_count = 32;
    }

    int problem_instances = thread_count;
    int inst_size = problem_instances * TPI;
    const int block_size = 512;
    int block_num = (inst_size / block_size) + (inst_size % block_size > 0 ? 1 : 0);

    const bool initial_factored = false;
    
    long B1, B2;
    cgbn_error_report_t *report;

    ChooseBounds(B1, B2, to_factor, DEFAULT_BOUNDS, 4); //Offset of 4 due to the use of the special parametirisation with theta
    EcmStart instance_local[problem_instances];
    EcmStart* instance_cuda;

    mpz_t theta, random_bound;
    mpz_inits(theta, random_bound, NULL);

    cgbn_mem_t<BITS>* mod_cuda;
    cgbn_mem_t<BITS> mod_local;
    long* primes_cuda;

    long seed = rand();
    gmp_randstate_t random_state;
    gmp_randinit_default(random_state);
    gmp_randseed_ui(random_state, seed);
    mpz_sub_ui(random_bound, to_factor, 7);

    CUDA_CHECK(hipSetDevice(0));
    bool is_factored = false;

    auto it = std::lower_bound(primes.begin(), primes.end(), B1);
    int copy_primes_count = std::distance(primes.begin(), it);

    from_mpz(to_factor, mod_local._limbs, BITS/32);

    CUDA_CHECK(hipMalloc(&instance_cuda, problem_instances*sizeof(EcmStart)));
    CUDA_CHECK(hipMalloc(&primes_cuda, copy_primes_count*sizeof(long)));
    CUDA_CHECK(hipMalloc(&mod_cuda, sizeof(cgbn_mem_t<BITS>)));
    CUDA_CHECK(cgbn_error_report_alloc(&report));

    while(!is_factored){
        for(int i = 0; i < problem_instances; i++){
            mpz_urandomm(theta, random_state, random_bound);
            mpz_add_ui(theta, theta, 6);

            MontgomeryCurve curve(to_factor);
            MontgomeryPoint starting_point;

            //Generate x, z, C from theta
            bool inverse_not_exist = curve.ComputeC_Q(theta, starting_point, output);
            if(inverse_not_exist){
               if(mpz_cmp(output, to_factor) == 0){ continue; }
                else{ return; }
            }

            from_mpz(starting_point.x, instance_local[i].x._limbs, BITS/32);
            from_mpz(starting_point.z, instance_local[i].z._limbs, BITS/32);
            from_mpz(curve.c, instance_local[i].c._limbs, BITS/32);
            from_mpz(curve.a_2_over_4, instance_local[i].a_2_over_4._limbs, BITS/32);

            instance_local[i].is_factored = false;
        }

        CUDA_CHECK(hipMemcpyToSymbol(HIP_SYMBOL(ecm_finished), &initial_factored, sizeof(bool)));
        CUDA_CHECK(hipMemcpy(instance_cuda, &instance_local,  problem_instances*sizeof(EcmStart), hipMemcpyHostToDevice));
        CUDA_CHECK(hipMemcpy(primes_cuda, primes.data(),  copy_primes_count*sizeof(long), hipMemcpyHostToDevice));
        CUDA_CHECK(hipMemcpy(mod_cuda, &mod_local, sizeof(cgbn_mem_t<BITS>), hipMemcpyHostToDevice));

        std::cout << "Block num " << block_num << " block size " << block_size << "\n";
        EcmKernel<<<block_num, block_size>>>(report, instance_cuda, inst_size, primes_cuda, mod_cuda, copy_primes_count, B1);
        CUDA_CHECK(hipDeviceSynchronize());
        CGBN_CHECK(report);

        hipMemcpy(&instance_local, instance_cuda, problem_instances*sizeof(EcmStart), hipMemcpyDeviceToHost);
       // std::cout << "Memory copied" << std::endl;
        for(int i = 0; i < problem_instances; i++){
            to_mpz(output, instance_local[i].c._limbs, BITS/32);
            //print_mpz("GCD: ", output);

            if(instance_local[i].is_factored){
                mpz_t x, z;
                mpz_inits(x, z, NULL);
                is_factored = true;
                 //The result is stored in the c coordinate of the starting point
                to_mpz(x, instance_local[i].x._limbs, BITS/32);
                to_mpz(z, instance_local[i].z._limbs, BITS/32);
                std::cout << "Finished factorising" << std::endl;
                print_mpz("Output: ", output);
                print_mpz("Point x:", x);
                print_mpz("Point z:", z);
                mpz_clears(x, z, NULL);
                break;
            }
        }
    }

    hipFree(&instance_cuda);
    hipFree(&primes_cuda);
    hipFree(&mod_cuda);

    mpz_clears(theta, random_bound, NULL);
}
