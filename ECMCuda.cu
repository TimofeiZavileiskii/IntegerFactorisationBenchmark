#include "hip/hip_runtime.h"
#include "PollardsRhoCuda.h"
#include "EllipticCurves.h"
#include "gmp.h"
#include <iostream>
#include "include/cgbn/cgbn.h"
#include "utility/cpu_support.h"
#include "utility/gpu_support.h"
#include "ECM.h"
#include <stdio.h>

#include <algorithm>

#define TPI 8
#if m_arith == 1
    #define BITS 32*8
#else
    #define BITS 32*16
#endif


#define BIT_COUNT_LONG(n) 64 - __clzll(n)
#define TEST_BIT(n, bit) ((n & (1 << bit)) != 0)

typedef cgbn_context_t<TPI> context_t;
typedef cgbn_env_t<context_t, BITS> env_t;


__device__ bool ecm_finished = false;


typedef struct{
    env_t::cgbn_t x;
    env_t::cgbn_t z;
    env_t::cgbn_t x_sub_z;
    env_t::cgbn_t x_add_z;
} PointCuda;


class CurveCuda{
    public:
    env_t::cgbn_t sqr1, sqr2, a_2_over_4, cross_1;
    env_t::cgbn_t mod, c;
    uint32_t montgomery_param;

    PointCuda point_copy;
    PointCuda point_double;
    
    __device__ __forceinline__ void ModularMul(env_t& env, env_t::cgbn_t& c, env_t::cgbn_t& a, env_t::cgbn_t& b){
        #if m_arith == 1
            cgbn_mont_mul(env, c, a, b, mod, montgomery_param);
        #else
            cgbn_mul(env, c, a, b);
            cgbn_rem(env, c, c, mod);
        #endif
    }

    __device__ __forceinline__ void ModularSqr(env_t& env, env_t::cgbn_t& c, env_t::cgbn_t& a){
        #if m_arith == 1
            cgbn_mont_sqr(env, c, a, mod, montgomery_param);
        #else
            cgbn_sqr(env, c, a);
            cgbn_rem(env, c, c, mod);
        #endif
    }

    __device__ __forceinline__ void ModularSub(env_t& env, env_t::cgbn_t& c, env_t::cgbn_t& a, env_t::cgbn_t& b){
        if(cgbn_compare(env, a, b) < 0){
            cgbn_add(env, c, a, mod);
            cgbn_sub(env, c, c, b);
        }
        else{
            cgbn_sub(env, c, a, b);
        }
    }

    __device__ __forceinline__ void ModularAdd(env_t& env, env_t::cgbn_t& c, env_t::cgbn_t& a, env_t::cgbn_t& b){
        cgbn_add(env, c, a, mod);
        if(cgbn_compare(env, c, mod) > 0){
            cgbn_sub(env, c, c, mod); 
        }
    }

    __device__ __forceinline__ void PointComputeDiffCuda(env_t& env, PointCuda& point){
        ModularSub(env, point.x_sub_z, point.x, point.z);
        ModularAdd(env, point.x_add_z, point.x, point.z);
    }

    __device__ void CopyPointCuda(env_t& env, PointCuda& point, PointCuda& point_copy){
        cgbn_set(env, point.x, point_copy.x);
        cgbn_set(env, point.z, point_copy.z);
        cgbn_set(env, point.x_add_z, point_copy.x_add_z);
        cgbn_set(env, point.x_sub_z, point_copy.x_sub_z);
    }

    /*
inline void DoublePoint(CurveCuda& curve, PointCuda& point){
        MontgomeryMul(sqr1, point.x_add_z, point.x_add_z, params);
        MontgomeryMul(sqr2, point.x_sub_z, point.x_sub_z, params);
        ModularSub(cross_1, sqr1, sqr2, params.mod);
        MontgomeryMul(point.x, sqr1, sqr2, params);
        MontgomeryMul(point.z, a_2_over_4, cross_1, params);
        ModularAdd(point.z, point.z, sqr2, params.mod);
        MontgomeryMul(point.z, point.z, cross_1, params);
        point.ComputeDiff(params.mod);
}*/

    __device__ void DoublePointCuda(env_t& env, PointCuda& point){
        ModularSqr(env, sqr1, point.x_add_z);
        ModularSqr(env, sqr2, point.x_sub_z);
        ModularSub(env, cross_1, sqr1, sqr2);
        ModularMul(env, point.x, sqr1, sqr2);
        ModularMul(env, point.z, a_2_over_4, cross_1);
        ModularAdd(env, point.z, point.z, sqr2);
        ModularMul(env, point.z, point.z, cross_1);
        PointComputeDiffCuda(env, point);
    }

    /*
    inline void AddPoints(MontgomeryPoint& point, MontgomeryPoint& to_add, MontgomeryPoint& p_min_add){
        if((mpz_cmp_ui(point.x, 0) == 0) && (mpz_cmp_ui(point.z, 0) == 0)){
            return;
        }
        if((mpz_cmp_ui(to_add.x, 0) == 0) && (mpz_cmp_ui(to_add.z, 0) == 0)){
            point.Copy(to_add);
            return;
        }

        #define cross_2 sqr1
        MontgomeryMul(cross_1, point.x_sub_z, to_add.x_add_z, params);
        MontgomeryMul(cross_2, point.x_add_z, to_add.x_sub_z, params);
        ModularAdd(to_add.x, cross_1, cross_2, params.mod);
        MontgomeryMul(to_add.x, to_add.x, to_add.x, params);
        MontgomeryMul(to_add.x, to_add.x, p_min_add.z, params);

        mpz_sub(to_add.z, cross_1, cross_2);
        MontgomeryMul(to_add.z, to_add.z, to_add.z, params);
        MontgomeryMul(to_add.z, to_add.z, p_min_add.x, params);
        to_add.ComputeDiff(params.mod);
        #undef cross_2
    }
*/

    __device__ void AddPointsCuda(env_t& env, PointCuda& point, PointCuda& to_add, PointCuda& diff){
        if((cgbn_compare_ui32(env, point.x, 0) == 0) && (cgbn_compare_ui32(env, point.z, 0) == 0)){
            return;
        }
        if((cgbn_compare_ui32(env, to_add.x, 0) == 0) && (cgbn_compare_ui32(env, to_add.z, 0) == 0)){
            CopyPointCuda(env, point, to_add);
            return;
        }

        #define cross_2 sqr1
        ModularMul(env, cross_1, point.x_sub_z, to_add.x_add_z);
        ModularMul(env, cross_2, point.x_add_z, to_add.x_sub_z);
        ModularAdd(env, to_add.x, cross_1, cross_2);

        ModularSqr(env, to_add.x, to_add.x);
        ModularMul(env, to_add.x, to_add.x, diff.z);

        ModularSub(env, to_add.z, cross_1, cross_2);
        ModularSqr(env, to_add.z, to_add.z);
        ModularMul(env, to_add.z, to_add.z, diff.x);
        PointComputeDiffCuda(env, to_add);
        #undef cross_2
    }

    /*
    inline void MultPoints(MontgomeryPoint& point, long multiple){
        if(multiple == 0){
            mpz_set_ui(point.x, 0);
            mpz_set_ui(point.z, 0);
            return;
        }
        if(multiple == 1){
            return;
        }
        if(multiple == 2){
            DoublePoint(point);
            return;
        }
        
        point.Copy(p_copy);
        point.Copy(p_double);
        DoublePoint(p_double);
        
        int bit_count = (int)log2((double)multiple)+1;

        for(int i = bit_count-2; i > 0; i--){
            if(test_bit(multiple, i) == 1){
                AddPoints(p_double, p_copy, point);
                DoublePoint(p_double);
            }
            else{
                AddPoints(p_copy, p_double, point);
                DoublePoint(p_copy);
            }
        }
        if(test_bit(multiple, 0) == 1){
            AddPoints(p_double, p_copy, point);
            p_copy.Copy(point);
        }
        else{
            DoublePoint(p_copy);
            p_copy.Copy(point);
        }
    }
    */

    __device__ __forceinline__ void MultPointCuda(env_t& env, PointCuda& point, long multiple){
        if(multiple == 0){
            cgbn_set_ui32(env, point.x, 0);
            cgbn_set_ui32(env, point.z, 0);
            return;
        }
        if(multiple == 1){
            return;
        }
        if(multiple == 2){
            DoublePointCuda(env, point);
            return;
        }

        CopyPointCuda(env, point, point_copy);
        CopyPointCuda(env, point, point_double);
        DoublePointCuda(env, point_double);

        int bit_count = BIT_COUNT_LONG(multiple);

        for(int i = bit_count-2; i > 0; i--){
            if(TEST_BIT(multiple, i)){
                AddPointsCuda(env, point_double, point_copy, point);
                DoublePointCuda(env, point_double);
            }
            else{
                AddPointsCuda(env, point_copy, point_double, point);
                DoublePointCuda(env, point_copy);
            }
            
        }

        if(TEST_BIT(multiple, 0)){
            AddPointsCuda(env, point_double, point_copy, point);
            CopyPointCuda(env, point_copy, point);
        }
        else{
            DoublePointCuda(env, point_copy);
            CopyPointCuda(env, point_copy, point);
        }
    }
};


typedef struct{
    cgbn_mem_t<BITS> c;
    cgbn_mem_t<BITS> x;
    cgbn_mem_t<BITS> z;
    cgbn_mem_t<BITS> a_2_over_4;
    bool is_factored;
} EcmStart;


__global__ void EcmKernel(EcmStart* start_instances, int inst_size, long* primes, cgbn_mem_t<BITS>* mod_host, int prime_count, long B1) {
    uint32_t instance = (blockIdx.x*blockDim.x + threadIdx.x) / TPI;
    if(instance >= inst_size){
        return;
    }

    context_t      bn_context(cgbn_no_checks);
    env_t          bn_env(bn_context.env<env_t>());
    env_t::cgbn_t gcd;

    CurveCuda curve;
    PointCuda point;

    cgbn_load(bn_env, curve.mod, mod_host);
    cgbn_load(bn_env, curve.c, &(start_instances[instance].c));
    cgbn_load(bn_env, point.x, &(start_instances[instance].x));
    cgbn_load(bn_env, point.z, &(start_instances[instance].z));
    cgbn_load(bn_env, curve.a_2_over_4, &(start_instances[instance].a_2_over_4));

    curve.montgomery_param = -cgbn_binary_inverse_ui32(bn_env, cgbn_get_ui32(bn_env, curve.mod));

    long e;

    float log_B1 = logf((float)B1);
    for(int index = 0; index < prime_count && !ecm_finished; index++){
        float prime = (float)primes[index];
        int repeat = (int)(log_B1/logf(prime));
        for(int rep = 0; rep < repeat; rep++){
            curve.MultPointCuda(bn_env, point, prime);}
    }

    cgbn_gcd(bn_env, gcd, curve.mod, point.z);
    if(cgbn_compare_ui32(bn_env, gcd, 1) > 0 && cgbn_compare(bn_env, gcd, curve.mod) < 0){
        
        start_instances[instance].is_factored = true;
        cgbn_store(bn_env, &start_instances[instance].x, point.x);
        cgbn_store(bn_env, &start_instances[instance].z, point.z);
        cgbn_store(bn_env, &start_instances[instance].c, gcd);
        __threadfence();
        ecm_finished = true;
    }
}

void EcmCuda(mpz_t output, mpz_t to_factor, int thread_count, const std::vector<long>& primes){
    if(thread_count == 0){
        thread_count = 2048;
    }

    int problem_instances = thread_count;
    int inst_size = problem_instances * TPI;
    const int block_size = 512;
    int block_num = (inst_size / block_size) + (inst_size % block_size > 0 ? 1 : 0);

    const bool initial_factored = false;
    
    long B1, B2;
    
    ChooseBounds(B1, B2, to_factor, DEFAULT_BOUNDS, 4); //Offset of 4 due to the use of the special parametirisation with theta
    EcmStart instance_local[problem_instances];
    EcmStart* instance_cuda;

    mpz_t theta, random_bound;
    mpz_inits(theta, random_bound, NULL);

    cgbn_mem_t<BITS>* mod_cuda;
    cgbn_mem_t<BITS> mod_local;
    long* primes_cuda;


    long seed = rand();
    gmp_randstate_t random_state;
    gmp_randinit_default(random_state);
    gmp_randseed_ui(random_state, seed);
    mpz_sub_ui(random_bound, to_factor, 7);

    CUDA_CHECK(hipSetDevice(0));
    bool is_factored = false;

    auto it = std::lower_bound(primes.begin(), primes.end(), B1);
    int copy_primes_count = std::distance(primes.begin(), it);

    CUDA_CHECK(hipMalloc(&instance_cuda, problem_instances*sizeof(EcmStart)));
    CUDA_CHECK(hipMalloc(&primes_cuda, copy_primes_count*sizeof(long)));
    CUDA_CHECK(hipMalloc(&mod_cuda, sizeof(cgbn_mem_t<BITS>)));

    while(!is_factored){
        for(int i = 0; i < problem_instances; i++){
            mpz_urandomm(theta, random_state, random_bound);
            mpz_add_ui(theta, theta, 6);

            MontgomeryCurve curve(to_factor);
            MontgomeryPoint starting_point;

            //Generate x, z, C from theta
            bool inverse_not_exist = curve.ComputeC_Q(theta, starting_point, output);
            if(inverse_not_exist){
                continue;
   //            if(mpz_cmp(output, to_factor) == 0){ continue; }
     //           else{ return; }
            }

            from_mpz(starting_point.x, instance_local[i].x._limbs, BITS/32);
            from_mpz(starting_point.z, instance_local[i].z._limbs, BITS/32);
            from_mpz(curve.c, instance_local[i].c._limbs, BITS/32);
            from_mpz(curve.a_2_over_4, instance_local[i].a_2_over_4._limbs, BITS/32);


            instance_local[i].is_factored = false;
        }

        CUDA_CHECK(hipMemcpyToSymbol(HIP_SYMBOL(ecm_finished), &initial_factored, sizeof(bool)));
        CUDA_CHECK(hipMemcpy(instance_cuda, &instance_local,  problem_instances*sizeof(EcmStart), hipMemcpyHostToDevice));
        CUDA_CHECK(hipMemcpy(primes_cuda, primes.data(),  copy_primes_count*sizeof(long), hipMemcpyHostToDevice));
        CUDA_CHECK(hipMemcpy(mod_cuda, &mod_local, sizeof(cgbn_mem_t<BITS>), hipMemcpyHostToDevice));

        std::cout << "Block num " << block_num << " block size " << block_size << "\n";
        EcmKernel<<<block_num, block_size>>>(instance_cuda, inst_size, primes_cuda, mod_cuda, copy_primes_count, B1);
        hipDeviceSynchronize(); // Wait for kernel to finish execution
        hipMemcpy(&instance_local, instance_cuda, problem_instances*sizeof(EcmStart), hipMemcpyDeviceToHost);
       // std::cout << "Memory copied" << std::endl;
        for(int i = 0; i < problem_instances; i++){
            if(instance_local[i].is_factored){
                mpz_t x, z;
                mpz_inits(x, z, NULL);
                is_factored = true;
                to_mpz(output, instance_local[i].c._limbs, BITS/32); //The result is stored in the c coordinate of the starting point
                to_mpz(x, instance_local[i].x._limbs, BITS/32);
                to_mpz(z, instance_local[i].z._limbs, BITS/32);
                std::cout << "Finished factorising" << std::endl;
                print_mpz("Output: ", output);
                print_mpz("Point x:", x);
                print_mpz("Point z:", z);
                mpz_clears(x, z, NULL);
                break;
            }
        }
    }

    hipFree(&instance_cuda);
    hipFree(&primes_cuda);
    hipFree(&mod_cuda);

    mpz_clears(theta, random_bound, NULL);
}
